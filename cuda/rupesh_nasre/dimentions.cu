#include <bits/stdc++.h>
#include <hip/hip_runtime.h>
__global__ void dkernel() {
    if (threadIdx.x == 0 && blockIdx.x == 0 &&
    threadIdx.y == 0 && blockIdx.y == 0 &&
    threadIdx.z == 0 && blockIdx.z == 0) {
        printf("%d %d %d %d %d %d.\nTIDs = %d %d %d\n----------------\n", gridDim.x, gridDim.y, gridDim.z, blockDim.x, blockDim.y, blockDim.z, threadIdx.x, threadIdx.y, threadIdx.z);
        // printf("", );
        // printf("");
    }
}

int main() {

    dim3 grid(2,3,4);
    dim3 block(5,6,7);

    dkernel<<<grid, block>>>();
    hipDeviceSynchronize();
    return 0;
}