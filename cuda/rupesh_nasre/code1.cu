#include <bits/stdc++.h>
#include <hip/hip_runtime.h>

__global__ void init(int *a, int ln) {
    unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < ln) {
        a[id] = id;
    }
}

int main() {
    int *da, N;
    N = 8000;
    int a[N];

    hipError_t err = hipMalloc(&da, N*sizeof(int));
    if (err != hipSuccess) {
        printf("hipMalloc failed: %s\n", hipGetErrorString(err));
        return -1;
    }
    // int no_of_blocks = ceil(N/1024);
    
    int threads_per_block = 1024;
    int no_of_blocks = (N + threads_per_block - 1) / threads_per_block;  // Ceiling division
    init<<<no_of_blocks, 1024>>>(da, N);
    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Kernel launch failed: %s\n", hipGetErrorString(err));
        return -1;
    }
    
    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        printf("hipDeviceSynchronize failed: %s\n", hipGetErrorString(err));
        return -1;
    }

    hipMemcpy(a, da, N*sizeof(int), hipMemcpyDeviceToHost);

    for (size_t i = 0; i < N; i++)
    {
        std::cout << a[i] << " ";
    }
    std::cout << "\n";
}