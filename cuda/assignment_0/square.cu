#include <bits/stdc++.h>
#include <hip/hip_runtime.h>

__global__ void sq_fun(int *dv_arr) {
    dv_arr[threadIdx.x] = dv_arr[threadIdx.x] * dv_arr[threadIdx.x];
    // std::cout << "printing from gpu: " << dv_arr[threadIdx.x] << std::endl;
    // printf("Hello from GPU\n");
    // printf("printing from gpu: %i \n", dv_arr[threadIdx.x]);
}

int main() {

    int N, *div_array;
    
    std::cin >> N;

    int v[N];
    for (size_t i = 0; i < N; i++) {
        std::cin >> v[i];
    }
    

    hipMalloc(&div_array, N * sizeof(int));
    hipMemcpy(div_array, v, N*sizeof(int), hipMemcpyHostToDevice);
    sq_fun<<<1, N>>>(div_array);
    // cudaDeviceSynchronize();
    hipMemcpy(v, div_array, N*sizeof(int), hipMemcpyDeviceToHost);
    // o/p from cpu:
    std::cout << "O/p after sqauring:\n";
    for (size_t i = 0; i < N; i++)
    {
        std::cout << v[i] << " ";
    }
    std::cout << std::endl;
}