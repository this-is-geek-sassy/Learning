#include <bits/stdc++.h>
#include <hip/hip_runtime.h>

__global__ void kernel() {
    printf("Hello, World from GPU!\n");
}

int main() {
    kernel<<<1, 1>>>();
    hipDeviceSynchronize();
    return 0;
}