#include <iostream>
#include <hip/hip_runtime.h>

// GPU kernel function
__global__ void vectorAdd(const float* a, const float* b, float* c, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        c[idx] = a[idx] + b[idx];
    }
}

int main() {
    int N = 10;
    size_t size = N * sizeof(float);

    // Allocate host memory
    float h_a[N], h_b[N], h_c[N];
    for (int i = 0; i < N; i++) {
        h_a[i] = i * 1.0f;
        h_b[i] = i * 2.0f;
    }

    // Allocate device memory
    float *d_a, *d_b, *d_c;
    hipMalloc(&d_a, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_c, size);

    // Copy data from host to device
    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);

    // Launch kernel: <<<numBlocks, threadsPerBlock>>>
    int threadsPerBlock = 256;
    int numBlocks = (N + threadsPerBlock - 1) / threadsPerBlock;
    vectorAdd<<<numBlocks, threadsPerBlock>>>(d_a, d_b, d_c, N);

    // Copy result back to host
    hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);

    // Print result
    std::cout << "Result: ";
    for (int i = 0; i < N; i++) {
        std::cout << h_c[i] << " ";
    }
    std::cout << std::endl;

    // Free device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}

